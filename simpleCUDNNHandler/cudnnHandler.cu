
#include <hip/hip_runtime.h>
#include <iostream>
#include "cudnn.h"

using namespace std;

int main(int argc, char const *argv[]) {
    cout << "hello cuda" << endl;
    cudnnStatus_t status;
    cudnnHandle_t handle;
    status = cudnnCreate(&handle);
    cout << "status " << status << endl; 
    cudnnDestroy(handle);
    return 0;
}
